#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

#define ARRAY_SIZE (1024 * 1024)

int main() {
    float *h_data = new float[ARRAY_SIZE];
    float *d_data;

    // 分配显存
    hipMalloc((void**)&d_data, ARRAY_SIZE * sizeof(float));

    // 记录开始时间
    auto start = std::chrono::high_resolution_clock::now();

    // 从内存复制数据到显存
    hipMemcpy(d_data, h_data, ARRAY_SIZE * sizeof(float), hipMemcpyHostToDevice);

    // 记录结束时间
    auto end = std::chrono::high_resolution_clock::now();

    // 计算传输时间
    std::chrono::duration<double> elapsed = end - start;

    // 计算传输速度 (GB/s)
    double transferSize = (double)ARRAY_SIZE * sizeof(float) / (1024 * 1024 * 1024);
    double transferSpeed = transferSize / elapsed.count();

    std::cout << "Transfer speed: " << transferSpeed << " GB/s" << std::endl;

    // 释放显存和内存
    hipFree(d_data);
    delete[] h_data;

    return 0;
}