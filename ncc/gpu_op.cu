#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define N 1000000000  // 向量和矩阵的大小（可根据需要调整）

// 点积
__global__ void dot_product_gpu(double *a, double *b, double *result) {
    __shared__ double cache[256];
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    double temp = 0.0;
    if (idx < N) {
        temp = a[idx] * b[idx];
    }

    int cacheIndex = tid;
    cache[cacheIndex] = temp;
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            cache[cacheIndex] += cache[cacheIndex + stride];
        }
        __syncthreads();
    }

    if (tid == 0) {
        atomicAdd(result, cache[0]);
    }
}

// 叉积（仅适用于三维向量）
__global__ void cross_product_gpu(double *a, double *b, double *result) {
    result[0] = a[1] * b[2] - a[2] * b[1];
    result[1] = a[2] * b[0] - a[0] * b[2];
    result[2] = a[0] * b[1] - a[1] * b[0];
}

// 外积
__global__ void outer_product_gpu(double *a, double *b, double *result) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N * N) {
        int row = idx / N;
        int col = idx % N;
        result[idx] = a[row] * b[col];
    }
}

// 矩阵乘积
__global__ void matrix_multiply_gpu(double *A, double *B, double *C) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        double sum = 0.0;
        for (int i = 0; i < N; i++) {
            sum += A[row * N + i] * B[i * N + col];
        }
        C[row * N + col] = sum;
    }
}

// Hadamard 积
__global__ void hadamard_product_gpu(double *A, double *B, double *C) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N * N) {
        C[idx] = A[idx] * B[idx];
    }
}

// Kronecker 积
__global__ void kronecker_product_gpu(double *A, double *B, double *C) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N * N * N * N) {
        int row = idx / (N * N);
        int col = idx % (N * N);
        int A_row = row / N;
        int A_col = col / N;
        int B_row = row % N;
        int B_col = col % N;
        C[idx] = A[A_row * N + A_col] * B[B_row * N + B_col];
    }
}

void measure_time(hipEvent_t start, hipEvent_t stop, float *elapsedTime) {
    hipEventRecord(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(elapsedTime, start, stop);
}

int main() {
    // 初始化随机数种子
    srand(time(NULL));

    // 分配主机内存
    double *a = (double *)malloc(N * sizeof(double));
    double *b = (double *)malloc(N * sizeof(double));
    double *result = (double *)malloc(sizeof(double));
    double *A = (double *)malloc(N * N * sizeof(double));
    double *B = (double *)malloc(N * N * sizeof(double));
    double *C = (double *)malloc(N * N * sizeof(double));

    // 初始化数据
    for (int i = 0; i < N; i++) {
        a[i] = (double)(rand() % 100);
        b[i] = (double)(rand() % 100);
    }
    for (int i = 0; i < N * N; i++) {
        A[i] = (double)(rand() % 100);
        B[i] = (double)(rand() % 100);
    }

    // 分配设备内存
    double *d_a, *d_b, *d_result, *d_A, *d_B, *d_C;
    hipMalloc(&d_a, N * sizeof(double));
    hipMalloc(&d_b, N * sizeof(double));
    hipMalloc(&d_result, sizeof(double));
    hipMalloc(&d_A, N * N * sizeof(double));
    hipMalloc(&d_B, N * N * sizeof(double));
    hipMalloc(&d_C, N * N * sizeof(double));

    // 创建 CUDA 事件
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // 点积
    hipMemcpy(d_a, a, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemset(d_result, 0, sizeof(double));

    hipEventRecord(start);
    dot_product_gpu<<<(N + 255) / 256, 256>>>(d_a, d_b, d_result);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float dotTime;
    hipEventElapsedTime(&dotTime, start, stop);

    hipMemcpy(result, d_result, sizeof(double), hipMemcpyDeviceToHost);
    printf("Dot Product Result: %f\n", result[0]);
    printf("Dot Product Time: %.3f ms\n", dotTime);

    // 叉积
    double c[3];
    double *d_c;
    hipMalloc(&d_c, 3 * sizeof(double));

    hipMemcpy(d_a, a, 3 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, 3 * sizeof(double), hipMemcpyHostToDevice);

    hipEventRecord(start);
    cross_product_gpu<<<1, 1>>>(d_a, d_b, d_c);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float crossTime;
    hipEventElapsedTime(&crossTime, start, stop);

    hipMemcpy(c, d_c, 3 * sizeof(double), hipMemcpyDeviceToHost);
    printf("Cross Product Result: (%f, %f, %f)\n", c[0], c[1], c[2]);
    printf("Cross Product Time: %.3f ms\n", crossTime);

    // 外积
    hipMemcpy(d_a, a, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(double), hipMemcpyHostToDevice);

    hipEventRecord(start);
    outer_product_gpu<<<(N * N + 255) / 256, 256>>>(d_a, d_b, d_C);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float outerTime;
    hipEventElapsedTime(&outerTime, start, stop);

    hipMemcpy(C, d_C, N * N * sizeof(double), hipMemcpyDeviceToHost);
    printf("Outer Product Time: %.3f ms\n", outerTime);

    // 矩阵乘积
    hipMemcpy(d_A, A, N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * N * sizeof(double), hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x, (N + blockSize.y - 1) / blockSize.y);

    hipEventRecord(start);
    matrix_multiply_gpu<<<gridSize, blockSize>>>(d_A, d_B, d_C);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float matrixTime;
    hipEventElapsedTime(&matrixTime, start, stop);

    hipMemcpy(C, d_C, N * N * sizeof(double), hipMemcpyDeviceToHost);
    printf("Matrix Multiply Time: %.3f ms\n", matrixTime);

    // Hadamard 积
    hipMemcpy(d_A, A, N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * N * sizeof(double), hipMemcpyHostToDevice);

    hipEventRecord(start);
    hadamard_product_gpu<<<(N * N + 255) / 256, 256>>>(d_A, d_B, d_C);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float hadamardTime;
    hipEventElapsedTime(&hadamardTime, start, stop);

    hipMemcpy(C, d_C, N * N * sizeof(double), hipMemcpyDeviceToHost);
    printf("Hadamard Product Time: %.3f ms\n", hadamardTime);

    // Kronecker 积
    double *d_K;
    hipMalloc(&d_K, N * N * N * N * sizeof(double));

    hipEventRecord(start);
    kronecker_product_gpu<<<(N * N * N * N + 255) / 256, 256>>>(d_A, d_B, d_K);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float kroneckerTime;
    hipEventElapsedTime(&kroneckerTime, start, stop);

    printf("Kronecker Product Time: %.3f ms\n", kroneckerTime);

    // 释放内存
    free(a);
    free(b);
    free(result);
    free(A);
    free(B);
    free(C);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_result);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_c);
    hipFree(d_K);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}