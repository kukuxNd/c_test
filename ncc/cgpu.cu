#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define N 10000000  // 向量大小，可以根据需要调整

// CPU版本的点积计算
double dot_product_cpu(double *a, double *b) {
    double result = 0.0;
    for (int i = 0; i < N; i++) {
        result += a[i] * b[i];
    }
    return result;
}

// GPU版本的点积计算
__global__ void dot_product_gpu(double *a, double *b, double *result) {
    __shared__ double cache[256];  // 使用共享内存进行归约
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    double temp = 0.0;
    if (idx < N) {
        temp = a[idx] * b[idx];
    }

    int cacheIndex = tid;
    cache[cacheIndex] = temp;
    __syncthreads();

    int i = blockDim.x / 2;
    while (i != 0) {
        if (tid < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
        i /= 2;
    }

    if (tid == 0) {
        atomicAdd(result, cache[0]);
    }
}

int main() {
    double *a, *b, result_cpu, result_gpu;
    double *d_a, *d_b, *d_result;

    // 初始化向量
    a = (double *)malloc(N * sizeof(double));
    b = (double *)malloc(N * sizeof(double));
    for (int i = 0; i < N; i++) {
        a[i] = (double)(rand() % 100);
        b[i] = (double)(rand() % 100);
    }

    // CPU计算
    clock_t start_cpu = clock();
    result_cpu = dot_product_cpu(a, b);
    clock_t end_cpu = clock();
    double cpu_time = (double)(end_cpu - start_cpu) / CLOCKS_PER_SEC;

    // GPU计算
    hipMalloc(&d_a, N * sizeof(double));
    hipMalloc(&d_b, N * sizeof(double));
    hipMalloc(&d_result, sizeof(double));
    hipMemcpy(d_a, a, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemset(d_result, 0, sizeof(double));

    // 创建CUDA事件
    hipEvent_t start, stop;
    float milliseconds;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // 记录开始事件
    hipEventRecord(start);

    // 执行GPU内核
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    dot_product_gpu<<<numBlocks, blockSize>>>(d_a, d_b, d_result);

    // 记录结束事件
    hipEventRecord(stop);

    // 等待内核完成
    hipEventSynchronize(stop);

    // 获取GPU执行时间
    hipEventElapsedTime(&milliseconds, start, stop);

    // 获取结果
    hipMemcpy(&result_gpu, d_result, sizeof(double), hipMemcpyDeviceToHost);

    // 输出结果和耗时
    printf("CPU Result: %f\n", result_cpu);
    printf("GPU Result: %f\n", result_gpu);
    printf("CPU Time: %f seconds\n", cpu_time);
    printf("GPU Time: %f ms\n", milliseconds);

    // 释放资源
    free(a);
    free(b);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_result);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}